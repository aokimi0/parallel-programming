#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <stdexcept>
#include <chrono>
#include <string>

#include <hip/hip_runtime.h>
#include "common.h"

std::vector<ll> multiply_ntt_serial(std::vector<ll> p1, std::vector<ll> p2, ll mod, ll primitive_root);

#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void pointwise_mult_kernel(ll* out, const ll* in1, const ll* in2, int n, ll mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = ((int128)in1[idx] * in2[idx]) % mod;
    }
}

__global__ void final_scaling_kernel(ll* a, int n, ll n_inv, ll mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = ((int128)a[idx] * n_inv) % mod;
    }
}

__global__ void ntt_kernel_naive(ll* a, const ll* twiddles, int len, int n, ll mod) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int butterfly_grp_idx = tidx / (len / 2);
    int butterfly_idx_in_grp = tidx % (len / 2);
    int i = butterfly_grp_idx * len + butterfly_idx_in_grp;

    if (i < n) {
        ll w = twiddles[butterfly_idx_in_grp];
        ll u = a[i];
        ll v = ((int128)a[i + len / 2] * w) % mod;
        a[i] = (u + v) % mod;
        a[i + len / 2] = (u - v + mod) % mod;
    }
}

template <typename Reducer>
__global__ void pointwise_mult_kernel_optimized(ll* out, const ll* in1, const ll* in2, int n, const Reducer reducer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = reducer.multiply(in1[idx], in2[idx]);
    }
}

template <typename Reducer>
__global__ void final_scaling_kernel_optimized(ll* a, int n, ll n_inv, const Reducer reducer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = reducer.multiply(a[idx], n_inv);
    }
}

template<typename Reducer>
__global__ void ntt_kernel_optimized(ll* a, const ll* twiddles, int len, int n, const Reducer reducer) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int butterfly_grp_idx = tidx / (len / 2);
    int butterfly_idx_in_grp = tidx % (len / 2);
    int i = butterfly_grp_idx * len + butterfly_idx_in_grp;
    
    if (i < n) {
        ll w = twiddles[butterfly_idx_in_grp];
        ll u = a[i];
        ll v = reducer.multiply(a[i + len / 2], w);
        
        ll sum = u + v;
        a[i] = (sum >= reducer.mod) ? (sum - reducer.mod) : sum;
        
        ll diff = u - v;
        a[i + len / 2] = (diff < 0) ? (diff + reducer.mod) : diff;
    }
}

template<typename Reducer>
__global__ void from_mont_kernel(ll* a, int n, const Reducer reducer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = reducer.from_mont(a[idx]);
    }
}

template<typename Reducer>
__global__ void to_mont_kernel(ll* a, int n, const Reducer reducer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        a[idx] = reducer.to_mont(a[idx]);
    }
}

void bit_reverse_copy(std::vector<ll>& a) {
    int n = a.size();
    for (int i = 1, j = 0; i < n; i++) {
        int bit = n >> 1;
        for (; j & bit; bit >>= 1) {
            j ^= bit;
        }
        j ^= bit;
        if (i < j) {
            std::swap(a[i], a[j]);
        }
    }
}

void ntt_gpu_naive(ll* d_a, int n, bool invert, ll mod, ll root) {
    int threads_per_block = 256;
    for (int len = 2; len <= n; len <<= 1) {
        std::vector<ll> h_twiddles(len / 2);
        ll wlen = power(root, (mod - 1) / len, mod);
        if (invert) wlen = modInverse(wlen, mod);
        
        h_twiddles[0] = 1;
        for (int j = 1; j < len / 2; j++) {
            h_twiddles[j] = ((int128)h_twiddles[j - 1] * wlen) % mod;
        }
        
        ll* d_twiddles;
        CUDA_CHECK(hipMalloc((void**)&d_twiddles, (len / 2) * sizeof(ll)));
        CUDA_CHECK(hipMemcpy(d_twiddles, h_twiddles.data(), (len / 2) * sizeof(ll), hipMemcpyHostToDevice));

        int num_threads = n / 2;
        int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
        ntt_kernel_naive<<<num_blocks, threads_per_block>>>(d_a, d_twiddles, len, n, mod);
        CUDA_CHECK(hipGetLastError());
        
        CUDA_CHECK(hipFree(d_twiddles));
    }
}

template<typename Reducer>
void ntt_gpu_optimized(ll* d_a, int n, bool invert, const Reducer& reducer) {
    int threads_per_block = 256;
    
    std::vector<ll> h_all_twiddles;
    size_t total_twiddles = 0;
    for (int len = 2; len <= n; len <<= 1) {
        total_twiddles += len / 2;
    }
    h_all_twiddles.reserve(total_twiddles);

    ll root = power(3, (reducer.mod - 1) / n, reducer.mod);
    if (invert) {
        root = modInverse(root, reducer.mod);
    }
    
    for (int len = 2; len <= n; len <<= 1) {
        ll wlen_base = power(root, n / len, reducer.mod);
        ll w = 1;
        for (int j = 0; j < len / 2; j++) {
            h_all_twiddles.push_back(w);
            w = ((int128)w * wlen_base) % reducer.mod;
        }
    }
    
    ll* d_all_twiddles;
    CUDA_CHECK(hipMalloc((void**)&d_all_twiddles, total_twiddles * sizeof(ll)));
    CUDA_CHECK(hipMemcpy(d_all_twiddles, h_all_twiddles.data(), total_twiddles * sizeof(ll), hipMemcpyHostToDevice));

    size_t twiddle_offset = 0;
    for (int len = 2; len <= n; len <<= 1) {
        ll* d_twiddles_stage = d_all_twiddles + twiddle_offset;
        
        int num_threads = n / 2;
        int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
        
        ntt_kernel_optimized<Reducer><<<num_blocks, threads_per_block>>>(d_a, d_twiddles_stage, len, n, reducer);
        CUDA_CHECK(hipGetLastError());
        
        twiddle_offset += len / 2;
    }

    CUDA_CHECK(hipFree(d_all_twiddles));
}

template<typename Reducer>
void ntt_gpu_montgomery(ll* d_a, int n, bool invert, const Reducer& reducer) {
    int threads_per_block = 256;
    
    std::vector<ll> h_all_twiddles;
    size_t total_twiddles = 0;
    for (int len = 2; len <= n; len <<= 1) {
        total_twiddles += len / 2;
    }
    h_all_twiddles.reserve(total_twiddles);

    ll root = power(3, (reducer.mod - 1) / n, reducer.mod);
    if (invert) {
        root = modInverse(root, reducer.mod);
    }
    
    for (int len = 2; len <= n; len <<= 1) {
        ll wlen_base = power(root, n / len, reducer.mod);
        ll w = 1;
        for (int j = 0; j < len / 2; j++) {
            h_all_twiddles.push_back(w);
            w = ((int128)w * wlen_base) % reducer.mod; 
        }
    }
    
    ll* d_all_twiddles;
    CUDA_CHECK(hipMalloc((void**)&d_all_twiddles, total_twiddles * sizeof(ll)));
    CUDA_CHECK(hipMemcpy(d_all_twiddles, h_all_twiddles.data(), total_twiddles * sizeof(ll), hipMemcpyHostToDevice));

    int num_blocks_twid = (total_twiddles + threads_per_block - 1) / threads_per_block;
    to_mont_kernel<Reducer><<<num_blocks_twid, threads_per_block>>>(d_all_twiddles, total_twiddles, reducer);
    CUDA_CHECK(hipGetLastError());

    size_t twiddle_offset = 0;
    for (int len = 2; len <= n; len <<= 1) {
        ll* d_twiddles_stage = d_all_twiddles + twiddle_offset;
        
        int num_threads = n / 2;
        int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
        
        ntt_kernel_optimized<Reducer><<<num_blocks, threads_per_block>>>(d_a, d_twiddles_stage, len, n, reducer);
        CUDA_CHECK(hipGetLastError());
        
        twiddle_offset += len / 2;
    }

    CUDA_CHECK(hipFree(d_all_twiddles));
}

std::vector<ll> multiply_ntt_gpu(
    std::vector<ll>& poly1, std::vector<ll>& poly2, 
    ll mod, ll primitive_root, const std::string& method) {
    
    int n1 = poly1.size();
    int n2 = poly2.size();
    if (n1 == 0 || n2 == 0) return {};
    
    int target_len = n1 + n2 - 1;
    int n = 1;
    while (n < target_len) n <<= 1;
    
    poly1.resize(n);
    poly2.resize(n);
    
    bit_reverse_copy(poly1);
    bit_reverse_copy(poly2);

    ll *d_p1, *d_p2;
    CUDA_CHECK(hipMalloc((void**)&d_p1, n * sizeof(ll)));
    CUDA_CHECK(hipMalloc((void**)&d_p2, n * sizeof(ll)));
    CUDA_CHECK(hipMemcpy(d_p1, poly1.data(), n * sizeof(ll), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_p2, poly2.data(), n * sizeof(ll), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    if (method == "barrett") {
        BarrettReducer br(mod);
        ntt_gpu_optimized<BarrettReducer>(d_p1, n, false, br);
        ntt_gpu_optimized<BarrettReducer>(d_p2, n, false, br);
        pointwise_mult_kernel_optimized<BarrettReducer><<<blocks, threads>>>(d_p1, d_p1, d_p2, n, br);
        ntt_gpu_optimized<BarrettReducer>(d_p1, n, true, br);
        ll n_inv = modInverse(n, mod);
        final_scaling_kernel_optimized<BarrettReducer><<<blocks, threads>>>(d_p1, n, n_inv, br);
    } else if (method == "montgomery") {
        MontgomeryReducer mr(mod);
        
        to_mont_kernel<MontgomeryReducer><<<blocks, threads>>>(d_p1, n, mr);
        to_mont_kernel<MontgomeryReducer><<<blocks, threads>>>(d_p2, n, mr);

        ntt_gpu_montgomery(d_p1, n, false, mr);
        ntt_gpu_montgomery(d_p2, n, false, mr);

        pointwise_mult_kernel_optimized<MontgomeryReducer><<<blocks, threads>>>(d_p1, d_p1, d_p2, n, mr);
        
        ntt_gpu_montgomery(d_p1, n, true, mr);
        
        ll n_inv = modInverse(n, mod);
        ll n_inv_mont = mr.to_mont(n_inv);
        final_scaling_kernel_optimized<MontgomeryReducer><<<blocks, threads>>>(d_p1, n, n_inv_mont, mr);

        from_mont_kernel<MontgomeryReducer><<<blocks, threads>>>(d_p1, n, mr);

    } else { 
        ntt_gpu_naive(d_p1, n, false, mod, primitive_root);
        ntt_gpu_naive(d_p2, n, false, mod, primitive_root);
        pointwise_mult_kernel<<<blocks, threads>>>(d_p1, d_p1, d_p2, n, mod);
        ntt_gpu_naive(d_p1, n, true, mod, primitive_root);
        ll n_inv = modInverse(n, mod);
        final_scaling_kernel<<<blocks, threads>>>(d_p1, n, n_inv, mod);
    }

    std::vector<ll> result(n);
    CUDA_CHECK(hipMemcpy(result.data(), d_p1, n * sizeof(ll), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_p1));
    CUDA_CHECK(hipFree(d_p2));

    result.resize(target_len);
    return result;
}

int main() {
    std::vector<ll> p1, p2;
    ll mod = 998244353;
    ll primitive_root = 3;

    read_input(p1, p2, "input.txt");

    if (p1.empty() || p2.empty()) {
        std::cerr << "Error: Input vectors are empty. Check input.txt." << std::endl;
        return 1;
    }
    std::cout << "Read " << p1.size() << " coefficients for each polynomial." << std::endl;

    auto start_serial = std::chrono::high_resolution_clock::now();
    std::vector<ll> res_serial = multiply_ntt_serial(p1, p2, mod, primitive_root);
    auto end_serial = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> serial_time = end_serial - start_serial;
    std::cout << "--- Performance Results ---" << std::endl;
    std::cout << "CPU Serial time: " << serial_time.count() << " ms" << std::endl;
    write_output("output_serial.txt", res_serial);

    int n = 1;
    while (n < p1.size() + p2.size()) n <<= 1;
    
    std::vector<ll> h_p1 = p1;
    std::vector<ll> h_p2 = p2;
    h_p1.resize(n);
    h_p2.resize(n);
    bit_reverse_copy(h_p1);
    bit_reverse_copy(h_p2);

    ll *d_p1, *d_p2, *d_res;
    CUDA_CHECK(hipMalloc((void**)&d_p1, n * sizeof(ll)));
    CUDA_CHECK(hipMalloc((void**)&d_p2, n * sizeof(ll)));
    CUDA_CHECK(hipMalloc((void**)&d_res, n * sizeof(ll)));

    CUDA_CHECK(hipMemcpy(d_p1, h_p1.data(), n * sizeof(ll), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_p2, h_p2.data(), n * sizeof(ll), hipMemcpyHostToDevice));

    int threads_per_block = 256;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    std::vector<std::string> methods = {"naive", "barrett", "montgomery"};
    for (const auto& method : methods) {
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        CUDA_CHECK(hipMemcpy(d_p1, h_p1.data(), n * sizeof(ll), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_p2, h_p2.data(), n * sizeof(ll), hipMemcpyHostToDevice));

        CUDA_CHECK(hipEventRecord(start));

        if (method == "naive") {
            ntt_gpu_naive(d_p1, n, false, mod, primitive_root);
            ntt_gpu_naive(d_p2, n, false, mod, primitive_root);
            pointwise_mult_kernel<<<num_blocks, threads_per_block>>>(d_res, d_p1, d_p2, n, mod);
            ntt_gpu_naive(d_res, n, true, mod, primitive_root);
            ll n_inv = modInverse(n, mod);
            final_scaling_kernel<<<num_blocks, threads_per_block>>>(d_res, n, n_inv, mod);
        } else if (method == "barrett") {
            BarrettReducer br(mod);
            ntt_gpu_optimized<BarrettReducer>(d_p1, n, false, br);
            ntt_gpu_optimized<BarrettReducer>(d_p2, n, false, br);
            pointwise_mult_kernel_optimized<BarrettReducer><<<num_blocks, threads_per_block>>>(d_res, d_p1, d_p2, n, br);
            ntt_gpu_optimized<BarrettReducer>(d_res, n, true, br);
            ll n_inv = modInverse(n, mod);
            final_scaling_kernel_optimized<BarrettReducer><<<num_blocks, threads_per_block>>>(d_res, n, n_inv, br);
        } else if (method == "montgomery") {
            MontgomeryReducer mr(mod);
            to_mont_kernel<MontgomeryReducer><<<num_blocks, threads_per_block>>>(d_p1, n, mr);
            to_mont_kernel<MontgomeryReducer><<<num_blocks, threads_per_block>>>(d_p2, n, mr);
            ntt_gpu_montgomery<MontgomeryReducer>(d_p1, n, false, mr);
            ntt_gpu_montgomery<MontgomeryReducer>(d_p2, n, false, mr);
            pointwise_mult_kernel_optimized<MontgomeryReducer><<<num_blocks, threads_per_block>>>(d_res, d_p1, d_p2, n, mr);
            ntt_gpu_montgomery<MontgomeryReducer>(d_res, n, true, mr);
            ll n_inv = modInverse(n, mod);
            ll n_inv_mont = mr.to_mont(n_inv);
            final_scaling_kernel_optimized<MontgomeryReducer><<<num_blocks, threads_per_block>>>(d_res, n, n_inv_mont, mr);
            from_mont_kernel<MontgomeryReducer><<<num_blocks, threads_per_block>>>(d_res, n, mr);
        }

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float gpu_time_ms = 0;
        CUDA_CHECK(hipEventElapsedTime(&gpu_time_ms, start, stop));

        std::vector<ll> res_gpu(n);
        CUDA_CHECK(hipMemcpy(res_gpu.data(), d_res, n * sizeof(ll), hipMemcpyDeviceToHost));
        res_gpu.resize(res_serial.size()); 

        std::cout << "GPU " << method << " kernel time: " << gpu_time_ms << " ms" << std::endl;
        std::cout << "  - Speedup vs CPU: " << serial_time.count() / gpu_time_ms << "x" << std::endl;
        
        std::string out_filename = "output_" + method + ".txt";
        write_output(out_filename, res_gpu);

        bool correct = (res_serial.size() == res_gpu.size());
        if(correct) {
            for(size_t i = 0; i < res_serial.size(); ++i) {
                if (res_serial[i] != res_gpu[i]) {
                    correct = false;
                    break;
                }
            }
        }
        std::cout << "  - Verification: " << (correct ? "PASSED" : "FAILED") << std::endl;

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    CUDA_CHECK(hipFree(d_p1));
    CUDA_CHECK(hipFree(d_p2));
    CUDA_CHECK(hipFree(d_res));

    return 0;
} 